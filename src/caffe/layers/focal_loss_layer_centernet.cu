#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/focal_loss_layer_centernet.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void focalSigmoidLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int batch, const int channels, const int height,
          const int width, Dtype* counts, float gamma, float alpha) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    /*
    const int fw = index % width;
    const int fh = (index / width) % height;
    const int fc = (index / width / height) % channels;
    const int fn = (index / width / height) / channels;
    const int dim = (fn * channels + fc) * height * width;
    const Dtype* label_slice = label + dim;
    const Dtype* prob_slice = prob_data + dim;
    */
    const Dtype label_a = label[index];
    const Dtype prob_a = prob_data[index];
    if( label_a == Dtype(1)){
      loss[index] = -log(max(prob_a, Dtype(FLT_MIN))) * powf(1 -prob_a, alpha);
      counts[index] = 1;
    }else if(label_a < Dtype(1)){
      loss[index] = -log(max(1 - prob_a, Dtype(FLT_MIN))) * powf(prob_a, alpha) *
                     powf(1 - label_a, gamma);
      counts[index] = 0;
    }
  }
}

template <typename Dtype>
void CenterNetfocalSigmoidWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* label = bottom[1]->gpu_data();
    const int nthreads = prob_.count();
    batch_ = bottom[0]->num();
    num_class_ = bottom[0]->channels();
    width_ = bottom[0]->width();
    height_ = bottom[0]->height();
    Dtype* loss_data = bottom[0]->mutable_gpu_diff();
    Dtype* counts = prob_.mutable_gpu_diff();
    focalSigmoidLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data,
        batch_, num_class_, height_, width_, counts, gamma_, alpha_);
    Dtype loss;
    caffe_gpu_asum(nthreads, loss_data, &loss);
    Dtype valid_count = -1;
    caffe_gpu_asum(nthreads, counts, &valid_count);
    Dtype normalizer = LossLayer<Dtype>::GetNormalizer(
        normalization_, 1, 1, valid_count);
    top[0]->mutable_cpu_data()[0] = loss / normalizer;
    #if 1
    if(iterations_%100 == 0){
        std::cout<<"forward batch_: "<<batch_<<", num_class: "<<num_class_
        <<", height: "<<height_ << ", width: " <<width_
        <<", normalizer: "<<normalizer
        <<", postive_count: "<< valid_count <<", class total_loss: "<<loss/ normalizer<<std::endl;
    }
    iterations_++;
    #endif
    if (top.size() == 2) {
      top[1]->ShareData(prob_);
    }
}

template <typename Dtype>
__global__ void focalSigmoidLossBackwardGPU(const int nthreads,
          const Dtype* label, const Dtype* prob_data, Dtype* bottom_diff, 
          const int batch, const int channels, const int height,
          const int width, Dtype* counts, float gamma, float alpha) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        /*
        const int fw = index % width;
        const int fh = (index / width) % height;
        const int fc = (index / width / height) % channels;
        const int fn = (index / width / height) / channels;
        const int dim = (fn * channels + fc) * height * width;
        const Dtype* label_slice = label + dim;
        const Dtype* prob_slice = prob_data + dim;
        */
        const Dtype label_a = label[index];
        const Dtype prob_a = prob_data[index];
        if(label_a == Dtype(1)){
            bottom_diff[index] = powf(1 - prob_a, alpha) * 
                                            (alpha * prob_a * log(max(prob_a, Dtype(FLT_MIN))) - (1 - prob_a));
            counts[index] = 1;
        }else if(label_a < Dtype(1)){
            bottom_diff[index] = powf(1 - label_a, gamma) * powf(prob_a, alpha) * 
                                            ( prob_a - alpha* (1 - prob_a) * log(max(1 - prob_a, Dtype(FLT_MIN))));
            counts[index] = 0;
        }
    }
}

template <typename Dtype>
void CenterNetfocalSigmoidWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    if (propagate_down[1]) {
        LOG(FATAL) << this->type()
                    <<
                    " Layer cannot backpropagate to label inputs.";
    }
    if (propagate_down[0]) {
        Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
        const Dtype* prob_data = prob_.gpu_data();
        const Dtype* label = bottom[1]->gpu_data();
        const int nthreads = bottom[0]->count();
        Dtype* counts = prob_.mutable_gpu_diff();
        batch_ = bottom[0]->num();
        num_class_ = bottom[0]->channels();
        width_ = bottom[0]->width();
        height_ = bottom[0]->height();
        focalSigmoidLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
            CAFFE_CUDA_NUM_THREADS>>>(nthreads, label, prob_data, bottom_diff,
            batch_, num_class_, height_, width_, counts, gamma_, alpha_);
    
        Dtype valid_count = -1;
        if (normalization_ == LossParameter_NormalizationMode_VALID) {
            caffe_gpu_asum(nthreads, counts, &valid_count);
        }
        Dtype normalizer = LossLayer<Dtype>::GetNormalizer(
            normalization_, 1, 1, valid_count);
        const Dtype loss_weight = top[0]->cpu_diff()[0] / normalizer;
        caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(CenterNetfocalSigmoidWithLossLayer);

}  // namespace caffe
